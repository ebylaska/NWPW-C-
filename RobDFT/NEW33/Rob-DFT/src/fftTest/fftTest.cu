#include <iostream>
using namespace std;

#include <cassert>

#include "thrust/host_vector.h"
#include "thrust/device_vector.h"

#include <hipfft/hipfft.h>

#ifndef REAL
#define REAL float
#endif

hipfftHandle fftPlanMany_C2R, fftPlanMany_R2C; hipfftHandle fftPlan_C2R, fftPlan_R2C;

template <typename Real>
void inline initFFTs(int *dim, int neall) {
  // Create a batched 3D plan
  if(sizeof(Real) == sizeof(float) ) {
    hipfftPlanMany(&fftPlanMany_C2R, 3, dim, NULL, 1, 0, NULL, 1, 0, HIPFFT_C2R,neall);
    hipfftPlanMany(&fftPlanMany_R2C, 3, dim, NULL, 1, 0, NULL, 1, 0, HIPFFT_R2C,neall);
    hipfftPlan3d(&fftPlan_C2R, dim[0], dim[1], dim[2], HIPFFT_C2R);
    hipfftPlan3d(&fftPlan_R2C, dim[0], dim[1], dim[2], HIPFFT_R2C);
  } else {
    hipfftPlanMany(&fftPlanMany_C2R, 3, dim, NULL, 1, 0, NULL, 1, 0, HIPFFT_Z2D,neall);
    hipfftPlanMany(&fftPlanMany_R2C, 3, dim, NULL, 1, 0, NULL, 1, 0, HIPFFT_D2Z,neall);
    hipfftPlan3d(&fftPlan_C2R, dim[0], dim[1], dim[2], HIPFFT_Z2D);
    hipfftPlan3d(&fftPlan_R2C, dim[0], dim[1], dim[2], HIPFFT_D2Z);
  }

#ifdef FFT_ALL
  cufftSetCompatibilityMode(fftPlanMany_C2R, CUFFT_COMPATIBILITY_FFTW_ALL);
  cufftSetCompatibilityMode(fftPlanMany_R2C, CUFFT_COMPATIBILITY_FFTW_ALL);
  cufftSetCompatibilityMode(fftPlan_C2R, CUFFT_COMPATIBILITY_FFTW_ALL);
  cufftSetCompatibilityMode(fftPlan_R2C, CUFFT_COMPATIBILITY_FFTW_ALL); 
#else
  //cufftSetCompatibilityMode(fftPlanMany_C2R, CUFFT_COMPATIBILITY_NATIVE);
  //cufftSetCompatibilityMode(fftPlanMany_R2C, CUFFT_COMPATIBILITY_NATIVE);
  //cufftSetCompatibilityMode(fftPlan_C2R, CUFFT_COMPATIBILITY_NATIVE);
  //cufftSetCompatibilityMode(fftPlan_R2C, CUFFT_COMPATIBILITY_NATIVE); 
#endif
}

inline void _FFTerror(int ret) {
  switch(ret) {
  case HIPFFT_SETUP_FAILED: cerr << "SETUP_FAILED" << endl; break;
  case HIPFFT_INVALID_PLAN: cerr << "INVALID_PLAN" << endl; break;
  case HIPFFT_INVALID_VALUE: cerr << "INVALID_VALUE" << endl; break;
  case HIPFFT_EXEC_FAILED: cerr << "EXEC_FAILED" << endl; break;
  default: cerr << "UNKNOWN ret code " << ret << endl;
  }
}

//template specialization to handle different data types (float,double) 
inline void crFFT_(hipfftHandle myFFTplan, float* A, float* B ) {
  int ret=hipfftExecC2R(myFFTplan, (hipfftComplex*)A,(hipfftReal*) B);
  
  if(ret != HIPFFT_SUCCESS) {
    cerr << "C2R FFT failed! ret code " << ret << endl; _FFTerror(ret); exit(1);
  }
}

inline void crFFT_(hipfftHandle myFFTplan, double* A, double* B ) {
  int ret = hipfftExecZ2D(myFFTplan, (hipfftDoubleComplex*)A,(hipfftDoubleReal*) B);
  
  if(ret != HIPFFT_SUCCESS) {
    cerr << "Z2D FFT failed! ret code " << ret << endl; _FFTerror(ret); exit(1);
  }
}

inline void rcFFT_(hipfftHandle myFFTplan, float* A, float* B ) {
  int ret = hipfftExecR2C(myFFTplan, (hipfftReal*)A,(hipfftComplex*) B);

  if(ret != HIPFFT_SUCCESS) {
    cerr << "C R2C FFT failed!" << endl; _FFTerror(ret); exit(1);
  }
}

inline void rcFFT_(hipfftHandle myFFTplan, double* A, double* B ) {
  int ret = hipfftExecD2Z(myFFTplan, (hipfftDoubleReal*)A,(hipfftDoubleComplex*) B);

  if(ret != HIPFFT_SUCCESS) {
    cerr << "D2Z FFT failed!" << endl; _FFTerror(ret); exit(1);
  }
}

template <typename Real> inline void rcFFT_neall( Real* A, Real* B ) { rcFFT_(fftPlanMany_R2C, A,B); }
template <typename Real> inline void crFFT_neall( Real* A, Real* B ) { crFFT_(fftPlanMany_C2R, A,B); }
template <typename Real> inline void rcFFT_single( Real* A, Real* B ) { rcFFT_(fftPlan_R2C, A,B); } 
template <typename Real> inline void crFFT_single( Real* A, Real* B ) { crFFT_(fftPlan_C2R, A,B); }

template <typename Real> static void test(int *dim, int neall) {
  int nfft = dim[0]*dim[1]*dim[2];
  int n2ft3d = (dim[0]+2)*dim[1]*dim[2];

  // fill the test data
  thrust::host_vector<Real> h_testData(n2ft3d*neall);
  thrust::host_vector<Real> h_testData1(n2ft3d*neall);

  for(int fft=0; fft < neall; fft++)
    for(int k=0; k < n2ft3d; k++)
      h_testData[k+fft*n2ft3d] = k;

  for(int fft=0; fft < neall; fft++)
    for(int k=0; k < dim[2]; k++)
      for(int j=0; j < dim[1]; j++) {
	h_testData[dim[0]    + j*(dim[0]+2) + k*(dim[0]+2)*dim[1]+fft*n2ft3d] = 0;
	h_testData[dim[0]+ 1 + j*(dim[0]+2) + k*(dim[0]+2)*dim[1]+fft*n2ft3d] = 0;
      }
  
  thrust::device_vector<Real> d_testData = h_testData;
  rcFFT_neall(thrust::raw_pointer_cast(&d_testData[0]),
	      thrust::raw_pointer_cast(&d_testData[0]));
  crFFT_neall(thrust::raw_pointer_cast(&d_testData[0]),
	      thrust::raw_pointer_cast(&d_testData[0]));
  h_testData1 = d_testData;

  for(int fft=0; fft < neall; fft++)
    for (int i=0; i<n2ft3d; ++i) {
      h_testData1[i+fft*n2ft3d] /= (Real)nfft;
      assert(h_testData1[i+fft*n2ft3d] == h_testData1[i+fft*n2ft3d]);
      //cout << "test data " << h_testData1[i+fft*n2ft3d] << " should be " 
	   //<< h_testData[i+fft*n2ft3d] << endl;
    }
}

main(int argc, char *argv[])
{
  if(argc < 5) {
    cerr << "Use dim[0] dim[1] dim[2] numberFFT" << endl;
    exit(1);
  }
  
  int dim[] = { atoi(argv[1]), atoi(argv[2]), atoi(argv[3])};
  int neall=atoi(argv[4]);

  cerr << "dim[0] = " << dim[0] << endl;
  cerr << "dim[1] = " << dim[1] << endl;
  cerr << "dim[2] = " << dim[2] << endl;
  cerr << "neall = " << neall << endl;
  cerr << "sizeof(REAL) is " << sizeof(REAL) << " bytes" << endl;
  
  initFFTs<REAL>(dim,neall);
  test<REAL>(dim, neall);
}

